#include "hip/hip_runtime.h"
#include "cuda_helper.h"
#include "cuda_vectors.h"
#include "cuda_vector_uint2x4.h"


#define CUBEHASH512_TPB80 640
#define CUBEHASH512_TPB64 640
#define CUBEHASH512_TPB64F 640

#pragma region inlines

static __device__ __host__ __forceinline__ void rrounds(uint32_t *x) {

    //#pragma unroll 2
    for (int r = 0; r < 16; r++) {

		/* "add x_0jklm into x_1jklmn modulo 2^32 rotate x_0jklm upwards by 7 bits" */
		x[16] = x[16] + x[ 0]; x[ 0] = ROTL32(x[ 0], 7);x[17] = x[17] + x[ 1];x[ 1] = ROTL32(x[ 1], 7);
		x[18] = x[18] + x[ 2]; x[ 2] = ROTL32(x[ 2], 7);x[19] = x[19] + x[ 3];x[ 3] = ROTL32(x[ 3], 7);
		x[20] = x[20] + x[ 4]; x[ 4] = ROTL32(x[ 4], 7);x[21] = x[21] + x[ 5];x[ 5] = ROTL32(x[ 5], 7);
		x[22] = x[22] + x[ 6]; x[ 6] = ROTL32(x[ 6], 7);x[23] = x[23] + x[ 7];x[ 7] = ROTL32(x[ 7], 7);
		x[24] = x[24] + x[ 8]; x[ 8] = ROTL32(x[ 8], 7);x[25] = x[25] + x[ 9];x[ 9] = ROTL32(x[ 9], 7);
		x[26] = x[26] + x[10]; x[10] = ROTL32(x[10], 7);x[27] = x[27] + x[11];x[11] = ROTL32(x[11], 7);
		x[28] = x[28] + x[12]; x[12] = ROTL32(x[12], 7);x[29] = x[29] + x[13];x[13] = ROTL32(x[13], 7);
		x[30] = x[30] + x[14]; x[14] = ROTL32(x[14], 7);x[31] = x[31] + x[15];x[15] = ROTL32(x[15], 7);
		/* "swap x_00klm with x_01klm" "xor x_1jklm into x_0jklm" */
		SWAP(x[0], x[8]); x[0] ^= x[16]; x[8] ^= x[24]; SWAP(x[1], x[9]); x[1] ^= x[17]; x[9] ^= x[25];
		SWAP(x[2], x[10]); x[2] ^= x[18]; x[10] ^= x[26]; SWAP(x[3], x[11]); x[3] ^= x[19]; x[11] ^= x[27];
		SWAP(x[4], x[12]); x[4] ^= x[20]; x[12] ^= x[28]; SWAP(x[5], x[13]); x[5] ^= x[21]; x[13] ^= x[29];
		SWAP(x[6], x[14]); x[6] ^= x[22]; x[14] ^= x[30]; SWAP(x[7], x[15]); x[7] ^= x[23]; x[15] ^= x[31];
		/* "swap x_1jk0m with x_1jk1m" */
		SWAP(x[16], x[18]); SWAP(x[17], x[19]); SWAP(x[20], x[22]); SWAP(x[21], x[23]); SWAP(x[24], x[26]); SWAP(x[25], x[27]); SWAP(x[28], x[30]); SWAP(x[29], x[31]);
		/* "add x_0jklm into x_1jklm modulo 2^32 rotate x_0jklm upwards by 11 bits" */
		x[16] = x[16] + x[ 0]; x[ 0] = ROTL32(x[ 0],11);x[17] = x[17] + x[ 1];x[ 1] = ROTL32(x[ 1],11);
		x[18] = x[18] + x[ 2]; x[ 2] = ROTL32(x[ 2],11);x[19] = x[19] + x[ 3];x[ 3] = ROTL32(x[ 3],11);
		x[20] = x[20] + x[ 4]; x[ 4] = ROTL32(x[ 4],11);x[21] = x[21] + x[ 5];x[ 5] = ROTL32(x[ 5],11);
		x[22] = x[22] + x[ 6]; x[ 6] = ROTL32(x[ 6],11);x[23] = x[23] + x[ 7];x[ 7] = ROTL32(x[ 7],11);
		x[24] = x[24] + x[ 8]; x[ 8] = ROTL32(x[ 8],11);x[25] = x[25] + x[ 9];x[ 9] = ROTL32(x[ 9],11);
		x[26] = x[26] + x[10]; x[10] = ROTL32(x[10],11);x[27] = x[27] + x[11];x[11] = ROTL32(x[11],11);
		x[28] = x[28] + x[12]; x[12] = ROTL32(x[12],11);x[29] = x[29] + x[13];x[13] = ROTL32(x[13],11);
		x[30] = x[30] + x[14]; x[14] = ROTL32(x[14],11);x[31] = x[31] + x[15];x[15] = ROTL32(x[15],11);
		/* "swap x_0j0lm with x_0j1lm"  "xor x_1jklm into x_0jklm"  */
		SWAP(x[0], x[4]); x[0] ^= x[16]; x[4] ^= x[20]; SWAP(x[1], x[5]); x[1] ^= x[17]; x[5] ^= x[21];
		SWAP(x[2], x[6]); x[2] ^= x[18]; x[6] ^= x[22]; SWAP(x[3], x[7]); x[3] ^= x[19]; x[7] ^= x[23];
		SWAP(x[8], x[12]); x[8] ^= x[24]; x[12] ^= x[28]; SWAP(x[9], x[13]); x[9] ^= x[25]; x[13] ^= x[29];
		SWAP(x[10], x[14]); x[10] ^= x[26]; x[14] ^= x[30]; SWAP(x[11], x[15]); x[11] ^= x[27]; x[15] ^= x[31];
		/* "swap x_1jkl0 with x_1jkl1" */
		SWAP(x[16], x[17]); SWAP(x[18], x[19]); SWAP(x[20], x[21]); SWAP(x[22], x[23]); SWAP(x[24], x[25]); SWAP(x[26], x[27]); SWAP(x[28], x[29]); SWAP(x[30], x[31]);

    }
}

#pragma endregion

#pragma region CubeHash512_64

__global__
__launch_bounds__(CUBEHASH512_TPB64, 2)
void cuda_base_cubehash512_gpu_hash_64(const uint32_t threads, uint32_t *g_hash) {

	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads) {

        uint32_t *Hash = (uint32_t*)&g_hash[thread << 4];

        uint32_t x[32] = {
			0x2AEA2A61, 0x50F494D4, 0x2D538B8B, 0x4167D83E,
			0x3FEE2313, 0xC701CF8C, 0xCC39968E, 0x50AC5695,
			0x4D42C787, 0xA647A8B3, 0x97CF0BEF, 0x825B4537,
			0xEEF864D2, 0xF22090C4, 0xD0E5CD33, 0xA23911AE,
			0xFCD398D9, 0x148FE485, 0x1B017BEF, 0xB6444532,
			0x6A536159, 0x2FF5781C, 0x91FA7934, 0x0DBADEA9,
			0xD65C8A2B, 0xA5A70E75, 0xB1C62456, 0xBC796576,
			0x1921C8F7, 0xE7989AF1, 0x7795D246, 0xD43E3B44
        };

		*(uint2x4*)&x[ 0] ^= __ldg4((uint2x4*)&Hash[0]);
	    rrounds(x);

		*(uint2x4*)&x[ 0] ^= __ldg4((uint2x4*)&Hash[8]);
	    rrounds(x);

		x[0] ^= 0x80;
		rrounds(x);

	    x[31] ^= 1;
	    #pragma unroll 10
	    for (int i = 0; i < 10; i++)
            rrounds(x);

		*(uint2x4*)&Hash[0] = *(uint2x4*)&x[0];
		*(uint2x4*)&Hash[8] = *(uint2x4*)&x[8];
	}
}

__host__
void cuda_base_cubehash512_cpu_hash_64(const uint32_t threads, uint32_t *d_hash) {

	const uint32_t threadsperblock = CUBEHASH512_TPB64;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	cuda_base_cubehash512_gpu_hash_64<<<grid, block>>>(threads, d_hash);
}

#pragma endregion

#pragma region CubeHash512_64_final

__global__
__launch_bounds__(CUBEHASH512_TPB64F, 2)
void cuda_base_cubehash512_gpu_hash_64f(const uint32_t threads, const uint32_t* __restrict__ g_hash, const uint32_t startNonce, uint32_t *resNonce, const uint64_t target) {

	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads) {

        const uint32_t *Hash = &g_hash[thread << 4];

        uint32_t x[32] = {
			0x2AEA2A61, 0x50F494D4, 0x2D538B8B, 0x4167D83E,
			0x3FEE2313, 0xC701CF8C, 0xCC39968E, 0x50AC5695,
			0x4D42C787, 0xA647A8B3, 0x97CF0BEF, 0x825B4537,
			0xEEF864D2, 0xF22090C4, 0xD0E5CD33, 0xA23911AE,
			0xFCD398D9, 0x148FE485, 0x1B017BEF, 0xB6444532,
			0x6A536159, 0x2FF5781C, 0x91FA7934, 0x0DBADEA9,
			0xD65C8A2B, 0xA5A70E75, 0xB1C62456, 0xBC796576,
			0x1921C8F7, 0xE7989AF1, 0x7795D246, 0xD43E3B44
        };

		*(uint2x4*)&x[ 0] ^= __ldg4((uint2x4*)&Hash[0]);
	    rrounds(x);

		*(uint2x4*)&x[ 0] ^= __ldg4((uint2x4*)&Hash[8]);
	    rrounds(x);

		x[0] ^= 0x80;
		rrounds(x);

	    x[31] ^= 1;
	    #pragma unroll 10
	    for (int i = 0; i < 10; i++)
            rrounds(x);

        uint64_t check = *(uint64_t*)&x[6];
		if (check <= target) {
			uint32_t tmp = atomicExch(&resNonce[0], startNonce + thread);
            if (tmp != UINT32_MAX)
				resNonce[1] = tmp;
		}
	}
}

__host__
void cuda_base_cubehash512_cpu_hash_64f(const uint32_t threads, const uint32_t *d_hash, const uint32_t startNonce, uint32_t *d_resNonce, const uint64_t target)
{
	const uint32_t threadsperblock = CUBEHASH512_TPB64F;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	cuda_base_cubehash512_gpu_hash_64f<<<grid, block>>>(threads, d_hash, startNonce, d_resNonce, target);
}

#pragma endregion


#pragma region CubeHash512_80

__constant__ static __align__(64) uint32_t precalc_x[32];
__constant__ static __align__(64) uint32_t c_message[3];

__host__
void cuda_base_cubehash512_setBlock_80(uint32_t* endiandata) {

    uint32_t x[32] = {
		0x2AEA2A61, 0x50F494D4, 0x2D538B8B, 0x4167D83E,
		0x3FEE2313, 0xC701CF8C, 0xCC39968E, 0x50AC5695,
		0x4D42C787, 0xA647A8B3, 0x97CF0BEF, 0x825B4537,
		0xEEF864D2, 0xF22090C4, 0xD0E5CD33, 0xA23911AE,
		0xFCD398D9, 0x148FE485, 0x1B017BEF, 0xB6444532,
		0x6A536159, 0x2FF5781C, 0x91FA7934, 0x0DBADEA9,
		0xD65C8A2B, 0xA5A70E75, 0xB1C62456, 0xBC796576,
		0x1921C8F7, 0xE7989AF1, 0x7795D246, 0xD43E3B44
    };

    AS_UINT4(&x[0]) ^= AS_UINT4(&endiandata[0]);
	AS_UINT4(&x[4]) ^= AS_UINT4(&endiandata[4]);
	rrounds(x);

	AS_UINT4(&x[0]) ^= AS_UINT4(&endiandata[8]);
	AS_UINT4(&x[4]) ^= AS_UINT4(&endiandata[12]);
	rrounds(x);

	uint32_t message[4];
    AS_UINT4(&message[0]) = AS_UINT4(&endiandata[16]);

	hipMemcpyToSymbol(HIP_SYMBOL(precalc_x), x, sizeof(precalc_x), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_message), message, sizeof(c_message), 0, hipMemcpyHostToDevice);
	//hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), endiandata, sizeof(c_PaddedMessage80), 0, hipMemcpyHostToDevice);
}

__global__
__launch_bounds__(CUBEHASH512_TPB80, 2)
void cuda_base_cubehash512_gpu_hash_80(const uint32_t threads, const uint32_t startNounce, uint32_t *g_outhash) {

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads) {

		uint32_t x[32];

        *(uint2x4*)&x[0] = __ldg4((uint2x4*)&precalc_x[0]);
		*(uint2x4*)&x[8] = __ldg4((uint2x4*)&precalc_x[8]);
        *(uint2x4*)&x[16] = __ldg4((uint2x4*)&precalc_x[16]);
		*(uint2x4*)&x[24] = __ldg4((uint2x4*)&precalc_x[24]);
        
        AS_UINT2(&x[0]) ^= __ldg((uint2*)&c_message[0]);
        x[2] ^= c_message[2];
        x[3] ^= cuda_swab32(startNounce + thread);
        x[4] ^= 0x80;
   	    rrounds(x);

	    x[31] ^= 1;
	    #pragma unroll 10
	    for (int i = 0; i < 10; i++)
            rrounds(x);

        uint32_t* outHash = &g_outhash[thread << 4];
		*(uint2x4*)&outHash[0] = *(uint2x4*)&x[0];
		*(uint2x4*)&outHash[8] = *(uint2x4*)&x[8];
	}
}

__host__
void cuda_base_cubehash512_cpu_hash_80(const uint32_t threads, const uint32_t startNounce, uint32_t *d_hash) {

	const uint32_t threadsperblock = CUBEHASH512_TPB80;
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	cuda_base_cubehash512_gpu_hash_80 <<<grid, block>>> (threads, startNounce, d_hash);
}

#pragma endregion
