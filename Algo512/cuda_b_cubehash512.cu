#include "hip/hip_runtime.h"
#include <cuda_helper.h>
#include <cuda_vectors.h>


#define CUBEHASH512_TPB64 640
#define CUBEHASH512_TPB64F 640

#pragma region inlines

__device__ __forceinline__
static void rrounds(uint32_t *x) {

    //#pragma unroll 2
    for (int r = 0; r < 16; r++) {

		/* "add x_0jklm into x_1jklmn modulo 2^32 rotate x_0jklm upwards by 7 bits" */
		x[16] = x[16] + x[ 0]; x[ 0] = ROTL32(x[ 0], 7);x[17] = x[17] + x[ 1];x[ 1] = ROTL32(x[ 1], 7);
		x[18] = x[18] + x[ 2]; x[ 2] = ROTL32(x[ 2], 7);x[19] = x[19] + x[ 3];x[ 3] = ROTL32(x[ 3], 7);
		x[20] = x[20] + x[ 4]; x[ 4] = ROTL32(x[ 4], 7);x[21] = x[21] + x[ 5];x[ 5] = ROTL32(x[ 5], 7);
		x[22] = x[22] + x[ 6]; x[ 6] = ROTL32(x[ 6], 7);x[23] = x[23] + x[ 7];x[ 7] = ROTL32(x[ 7], 7);
		x[24] = x[24] + x[ 8]; x[ 8] = ROTL32(x[ 8], 7);x[25] = x[25] + x[ 9];x[ 9] = ROTL32(x[ 9], 7);
		x[26] = x[26] + x[10]; x[10] = ROTL32(x[10], 7);x[27] = x[27] + x[11];x[11] = ROTL32(x[11], 7);
		x[28] = x[28] + x[12]; x[12] = ROTL32(x[12], 7);x[29] = x[29] + x[13];x[13] = ROTL32(x[13], 7);
		x[30] = x[30] + x[14]; x[14] = ROTL32(x[14], 7);x[31] = x[31] + x[15];x[15] = ROTL32(x[15], 7);
		/* "swap x_00klm with x_01klm" "xor x_1jklm into x_0jklm" */
		xchg(x[0], x[8]); x[0] ^= x[16]; x[8] ^= x[24]; xchg(x[1], x[9]); x[1] ^= x[17]; x[9] ^= x[25];
		xchg(x[2], x[10]); x[2] ^= x[18]; x[10] ^= x[26]; xchg(x[3], x[11]); x[3] ^= x[19]; x[11] ^= x[27];
		xchg(x[4], x[12]); x[4] ^= x[20]; x[12] ^= x[28]; xchg(x[5], x[13]); x[5] ^= x[21]; x[13] ^= x[29];
		xchg(x[6], x[14]); x[6] ^= x[22]; x[14] ^= x[30]; xchg(x[7], x[15]); x[7] ^= x[23]; x[15] ^= x[31];
		/* "swap x_1jk0m with x_1jk1m" */
		xchg(x[16], x[18]); xchg(x[17], x[19]); xchg(x[20], x[22]); xchg(x[21], x[23]); xchg(x[24], x[26]); xchg(x[25], x[27]); xchg(x[28], x[30]); xchg(x[29], x[31]);
		/* "add x_0jklm into x_1jklm modulo 2^32 rotate x_0jklm upwards by 11 bits" */
		x[16] = x[16] + x[ 0]; x[ 0] = ROTL32(x[ 0],11);x[17] = x[17] + x[ 1];x[ 1] = ROTL32(x[ 1],11);
		x[18] = x[18] + x[ 2]; x[ 2] = ROTL32(x[ 2],11);x[19] = x[19] + x[ 3];x[ 3] = ROTL32(x[ 3],11);
		x[20] = x[20] + x[ 4]; x[ 4] = ROTL32(x[ 4],11);x[21] = x[21] + x[ 5];x[ 5] = ROTL32(x[ 5],11);
		x[22] = x[22] + x[ 6]; x[ 6] = ROTL32(x[ 6],11);x[23] = x[23] + x[ 7];x[ 7] = ROTL32(x[ 7],11);
		x[24] = x[24] + x[ 8]; x[ 8] = ROTL32(x[ 8],11);x[25] = x[25] + x[ 9];x[ 9] = ROTL32(x[ 9],11);
		x[26] = x[26] + x[10]; x[10] = ROTL32(x[10],11);x[27] = x[27] + x[11];x[11] = ROTL32(x[11],11);
		x[28] = x[28] + x[12]; x[12] = ROTL32(x[12],11);x[29] = x[29] + x[13];x[13] = ROTL32(x[13],11);
		x[30] = x[30] + x[14]; x[14] = ROTL32(x[14],11);x[31] = x[31] + x[15];x[15] = ROTL32(x[15],11);
		/* "swap x_0j0lm with x_0j1lm"  "xor x_1jklm into x_0jklm"  */
		xchg(x[0], x[4]); x[0] ^= x[16]; x[4] ^= x[20]; xchg(x[1], x[5]); x[1] ^= x[17]; x[5] ^= x[21];
		xchg(x[2], x[6]); x[2] ^= x[18]; x[6] ^= x[22]; xchg(x[3], x[7]); x[3] ^= x[19]; x[7] ^= x[23];
		xchg(x[8], x[12]); x[8] ^= x[24]; x[12] ^= x[28]; xchg(x[9], x[13]); x[9] ^= x[25]; x[13] ^= x[29];
		xchg(x[10], x[14]); x[10] ^= x[26]; x[14] ^= x[30]; xchg(x[11], x[15]); x[11] ^= x[27]; x[15] ^= x[31];
		/* "swap x_1jkl0 with x_1jkl1" */
		xchg(x[16], x[17]); xchg(x[18], x[19]); xchg(x[20], x[21]); xchg(x[22], x[23]); xchg(x[24], x[25]); xchg(x[26], x[27]); xchg(x[28], x[29]); xchg(x[30], x[31]);

    }
}

#pragma endregion

#pragma region CubeHash512_64

__global__
__launch_bounds__(CUBEHASH512_TPB64, 2)
void cuda_base_cubehash512_gpu_hash_64(const uint32_t threads, uint32_t *g_hash) {

	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads) {

        uint32_t *Hash = (uint32_t*)&g_hash[thread << 4];

        uint32_t x[32] = {
			0x2AEA2A61, 0x50F494D4, 0x2D538B8B, 0x4167D83E,
			0x3FEE2313, 0xC701CF8C, 0xCC39968E, 0x50AC5695,
			0x4D42C787, 0xA647A8B3, 0x97CF0BEF, 0x825B4537,
			0xEEF864D2, 0xF22090C4, 0xD0E5CD33, 0xA23911AE,
			0xFCD398D9, 0x148FE485, 0x1B017BEF, 0xB6444532,
			0x6A536159, 0x2FF5781C, 0x91FA7934, 0x0DBADEA9,
			0xD65C8A2B, 0xA5A70E75, 0xB1C62456, 0xBC796576,
			0x1921C8F7, 0xE7989AF1, 0x7795D246, 0xD43E3B44
        };

        AS_UINT4(&x[0]) ^= AS_UINT4(&Hash[0]);
	    AS_UINT4(&x[4]) ^= AS_UINT4(&Hash[4]);
	    rrounds(x);

	    AS_UINT4(&x[0]) ^= AS_UINT4(&Hash[8]);
	    AS_UINT4(&x[4]) ^= AS_UINT4(&Hash[12]);
	    rrounds(x);

		x[0] ^= 0x80;
		rrounds(x);

	    x[31] ^= 1;
	    #pragma unroll 10
	    for (int i = 0; i < 10; i++)
            rrounds(x);

	    AS_UINT4(&Hash[ 0]) = AS_UINT4(&x[0]);
	    AS_UINT4(&Hash[ 4]) = AS_UINT4(&x[4]);
	    AS_UINT4(&Hash[ 8]) = AS_UINT4(&x[8]);
	    AS_UINT4(&Hash[12]) = AS_UINT4(&x[12]);
	}
}

__host__
void cuda_base_cubehash512_cpu_hash_64(const uint32_t threads, uint32_t *d_hash)
{
	const uint32_t threadsperblock = CUBEHASH512_TPB64;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	cuda_base_cubehash512_gpu_hash_64<<<grid, block>>>(threads, d_hash);
}

#pragma endregion

#pragma region CubeHash512_64_final

__global__
__launch_bounds__(CUBEHASH512_TPB64F, 2)
void cuda_base_cubehash512_gpu_hash_64f(const uint32_t threads, const uint32_t* __restrict__ g_hash, const uint32_t startNonce, uint32_t *resNonce, const uint64_t target) {

	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads) {

        const uint32_t *Hash = &g_hash[thread << 4];

        uint32_t x[32] = {
			0x2AEA2A61, 0x50F494D4, 0x2D538B8B, 0x4167D83E,
			0x3FEE2313, 0xC701CF8C, 0xCC39968E, 0x50AC5695,
			0x4D42C787, 0xA647A8B3, 0x97CF0BEF, 0x825B4537,
			0xEEF864D2, 0xF22090C4, 0xD0E5CD33, 0xA23911AE,
			0xFCD398D9, 0x148FE485, 0x1B017BEF, 0xB6444532,
			0x6A536159, 0x2FF5781C, 0x91FA7934, 0x0DBADEA9,
			0xD65C8A2B, 0xA5A70E75, 0xB1C62456, 0xBC796576,
			0x1921C8F7, 0xE7989AF1, 0x7795D246, 0xD43E3B44
        };

        AS_UINT4(&x[0]) ^= AS_UINT4(&Hash[0]);
	    AS_UINT4(&x[4]) ^= AS_UINT4(&Hash[4]);
	    rrounds(x);

	    AS_UINT4(&x[0]) ^= AS_UINT4(&Hash[8]);
	    AS_UINT4(&x[4]) ^= AS_UINT4(&Hash[12]);
	    rrounds(x);

		x[0] ^= 0x80;
		rrounds(x);

	    x[31] ^= 1;
	    #pragma unroll 10
	    for (int i = 0; i < 10; i++)
            rrounds(x);

        uint64_t check = *(uint64_t*)&x[6];
		if (check <= target) {
			uint32_t tmp = atomicExch(&resNonce[0], startNonce + thread);
            if (tmp != UINT32_MAX)
				resNonce[1] = tmp;
		}
	}
}

__host__
void cuda_base_cubehash512_cpu_hash_64f(const uint32_t threads, const uint32_t *d_hash, const uint32_t startNonce, uint32_t *d_resNonce, const uint64_t target)
{
	const uint32_t threadsperblock = CUBEHASH512_TPB64F;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	cuda_base_cubehash512_gpu_hash_64f<<<grid, block>>>(threads, d_hash, startNonce, d_resNonce, target);
}

#pragma endregion


//#define WANT_CUBEHASH80
#ifdef WANT_CUBEHASH80

__constant__
static uint32_t c_PaddedMessage80[20];

__host__
void cubehash512_setBlock_80(int thr_id, uint32_t* endiandata)
{
	hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), endiandata, sizeof(c_PaddedMessage80), 0, hipMemcpyHostToDevice);
}

__global__
void cubehash512_gpu_hash_80(const uint32_t threads, const uint32_t startNounce, uint64_t *g_outhash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nonce = startNounce + thread;

		uint32_t x[2][2][2][2][2];
		Init(x);

		uint32_t message[8];
		// first 32 bytes
		AS_UINT4(&message[0]) = AS_UINT4(&c_PaddedMessage80[0]);
		AS_UINT4(&message[4]) = AS_UINT4(&c_PaddedMessage80[4]);
		Update32(x, message);

		// second 32 bytes
		AS_UINT4(&message[0]) = AS_UINT4(&c_PaddedMessage80[8]);
		AS_UINT4(&message[4]) = AS_UINT4(&c_PaddedMessage80[12]);
		Update32(x, message);

		// last 16 bytes + Padding
		AS_UINT4(&message[0]) = AS_UINT4(&c_PaddedMessage80[16]);
		message[3] = cuda_swab32(nonce);
		message[4] = 0x80;
		message[5] = 0;
		message[6] = 0;
		message[7] = 0;
		Update32(x, message);

		uint32_t* output = (uint32_t*) (&g_outhash[(size_t)8 * thread]);
		Final(x, output);
	}
}

__host__
void cubehash512_cuda_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNounce, uint32_t *d_hash)
{
	const uint32_t threadsperblock = 256;
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	cubehash512_gpu_hash_80 <<<grid, block>>> (threads, startNounce, (uint64_t*) d_hash);
}

#endif