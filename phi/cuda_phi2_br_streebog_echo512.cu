#include "hip/hip_runtime.h"
/**
* Streebog + Echo512 branch cuda kernel for Phi2
* 
* DumaxFr@github 2018
*/

#include <stdio.h>

#include "miner.h"
#include "cuda_helper.h"
#include "cuda_vector_uint2x4.h"

#define INTENSIVE_GMF
#include "aes/cuda_aes.cuh"

#include "x11/streebog_arrays.cuh"

#ifdef __INTELLISENSE__
extern __device__ __device_builtin__ void __threadfence_block(void);
#endif // __INTELLISENSE__


#define PHI2BRANCH_TPB64 128

#pragma region Streebog_macros

#if PHI2BRANCH_TPB64 == 128
// shared[0][] & shared[7][] are always __ldg'ed
#define LUT_GPU_INIT(sharedMemory, idx) { \
	/*sharedMemory[0][(idx<<1) + 0] = __ldg(&T02[(idx<<1) + 0]);*/ \
	/*sharedMemory[0][(idx<<1) + 1] = __ldg(&T02[(idx<<1) + 1]);*/ \
	sharedMemory[0][(idx<<1) + 0] = __ldg(&T12[(idx<<1) + 0]); \
	sharedMemory[0][(idx<<1) + 1] = __ldg(&T12[(idx<<1) + 1]); \
	sharedMemory[1][(idx<<1) + 0] = __ldg(&T22[(idx<<1) + 0]); \
	sharedMemory[1][(idx<<1) + 1] = __ldg(&T22[(idx<<1) + 1]); \
	sharedMemory[2][(idx<<1) + 0] = __ldg(&T32[(idx<<1) + 0]); \
	sharedMemory[2][(idx<<1) + 1] = __ldg(&T32[(idx<<1) + 1]); \
	sharedMemory[3][(idx<<1) + 0] = __ldg(&T42[(idx<<1) + 0]); \
	sharedMemory[3][(idx<<1) + 1] = __ldg(&T42[(idx<<1) + 1]); \
	sharedMemory[4][(idx<<1) + 0] = __ldg(&T52[(idx<<1) + 0]); \
	sharedMemory[4][(idx<<1) + 1] = __ldg(&T52[(idx<<1) + 1]); \
	sharedMemory[5][(idx<<1) + 0] = __ldg(&T62[(idx<<1) + 0]); \
	sharedMemory[5][(idx<<1) + 1] = __ldg(&T62[(idx<<1) + 1]); \
	/*sharedMemory[7][(idx<<1) + 0] = __ldg(&T72[(idx<<1) + 0]);*/ \
	/*sharedMemory[7][(idx<<1) + 1] = __ldg(&T72[(idx<<1) + 1]);*/ \
}
#elif PHI2BRANCH_TPB64 == 256
#define LUT_GPU_INIT(sharedMemory, idx) { \
	/*sharedMemory[0][idx] = __ldg(&T02[idx]);*/ \
	sharedMemory[0][idx] = __ldg(&T12[idx]); \
	sharedMemory[1][idx] = __ldg(&T22[idx]); \
	sharedMemory[2][idx] = __ldg(&T32[idx]); \
	sharedMemory[3][idx] = __ldg(&T42[idx]); \
	sharedMemory[4][idx] = __ldg(&T52[idx]); \
	sharedMemory[5][idx] = __ldg(&T62[idx]); \
	/*sharedMemory[7][idx] = __ldg(&T72[idx]);*/ \
}
#elif PHI2BRANCH_TPB64 < 256
#define LUT_GPU_INIT(sharedMemory, idx) { \
    if (idx < 128) { \
	    /*sharedMemory[0][(idx<<1) + 0] = __ldg(&T02[(idx<<1) + 0]);*/ \
	    /*sharedMemory[0][(idx<<1) + 1] = __ldg(&T02[(idx<<1) + 1]);*/ \
	    sharedMemory[0][(idx<<1) + 0] = __ldg(&T12[(idx<<1) + 0]); \
	    sharedMemory[0][(idx<<1) + 1] = __ldg(&T12[(idx<<1) + 1]); \
	    sharedMemory[1][(idx<<1) + 0] = __ldg(&T22[(idx<<1) + 0]); \
	    sharedMemory[1][(idx<<1) + 1] = __ldg(&T22[(idx<<1) + 1]); \
	    sharedMemory[2][(idx<<1) + 0] = __ldg(&T32[(idx<<1) + 0]); \
	    sharedMemory[2][(idx<<1) + 1] = __ldg(&T32[(idx<<1) + 1]); \
	    sharedMemory[3][(idx<<1) + 0] = __ldg(&T42[(idx<<1) + 0]); \
	    sharedMemory[3][(idx<<1) + 1] = __ldg(&T42[(idx<<1) + 1]); \
	    sharedMemory[4][(idx<<1) + 0] = __ldg(&T52[(idx<<1) + 0]); \
	    sharedMemory[4][(idx<<1) + 1] = __ldg(&T52[(idx<<1) + 1]); \
	    sharedMemory[5][(idx<<1) + 0] = __ldg(&T62[(idx<<1) + 0]); \
	    sharedMemory[5][(idx<<1) + 1] = __ldg(&T62[(idx<<1) + 1]); \
	    /*sharedMemory[7][(idx<<1) + 0] = __ldg(&T72[(idx<<1) + 0]);*/ \
	    /*sharedMemory[7][(idx<<1) + 1] = __ldg(&T72[(idx<<1) + 1]);*/ \
    } \
}
#else
#define LUT_GPU_INIT(sharedMemory, idx) { \
    if (idx < 256) { \
	    /*sharedMemory[0][idx] = __ldg(&T02[idx]);*/ \
	    sharedMemory[0][idx] = __ldg(&T12[idx]); \
	    sharedMemory[1][idx] = __ldg(&T22[idx]); \
	    sharedMemory[2][idx] = __ldg(&T32[idx]); \
	    sharedMemory[3][idx] = __ldg(&T42[idx]); \
	    sharedMemory[4][idx] = __ldg(&T52[idx]); \
	    sharedMemory[5][idx] = __ldg(&T62[idx]); \
	    /*sharedMemory[7][idx] = __ldg(&T72[idx]);*/ \
    } \
}
#endif

#pragma endregion


#pragma region Streebog_inlines
__device__ __forceinline__
static void GOST_FS(const uint2 shared[6][256], const uint2* const __restrict__ state, uint2* return_state) {

	return_state[0] = __ldg(&T02[__byte_perm(state[7].x,0,0x44440)])
			^ shared[0][__byte_perm(state[6].x,0,0x44440)]
			^ shared[1][__byte_perm(state[5].x,0,0x44440)]
			^ shared[2][__byte_perm(state[4].x,0,0x44440)]
			^ shared[3][__byte_perm(state[3].x,0,0x44440)]
			^ shared[4][__byte_perm(state[2].x,0,0x44440)]
			^ shared[5][__byte_perm(state[1].x,0,0x44440)]
			^ __ldg(&T72[__byte_perm(state[0].x,0,0x44440)]);

	return_state[1] = __ldg(&T02[__byte_perm(state[7].x,0,0x44441)])
			^ __ldg(&T12[__byte_perm(state[6].x,0,0x44441)])
			^ shared[1][__byte_perm(state[5].x,0,0x44441)]
			^ shared[2][__byte_perm(state[4].x,0,0x44441)]
			^ shared[3][__byte_perm(state[3].x,0,0x44441)]
			^ shared[4][__byte_perm(state[2].x,0,0x44441)]
			^ shared[5][__byte_perm(state[1].x,0,0x44441)]
			^ __ldg(&T72[__byte_perm(state[0].x,0,0x44441)]);

	return_state[2] = __ldg(&T02[__byte_perm(state[7].x,0,0x44442)])
			^ __ldg(&T12[__byte_perm(state[6].x,0,0x44442)])
			^ shared[1][__byte_perm(state[5].x,0,0x44442)]
			^ shared[2][__byte_perm(state[4].x,0,0x44442)]
			^ shared[3][__byte_perm(state[3].x,0,0x44442)]
			^ shared[4][__byte_perm(state[2].x,0,0x44442)]
			^ __ldg(&T72[__byte_perm(state[0].x,0,0x44442)])
			^ shared[5][__byte_perm(state[1].x,0,0x44442)];

	return_state[3] = __ldg(&T02[__byte_perm(state[7].x,0,0x44443)])
			^ shared[0][__byte_perm(state[6].x,0,0x44443)]
			^ shared[1][__byte_perm(state[5].x,0,0x44443)]
			^ shared[2][__byte_perm(state[4].x,0,0x44443)]
			^ __ldg(&T42[__byte_perm(state[3].x,0,0x44443)])
			^ shared[4][__byte_perm(state[2].x,0,0x44443)]
			^ __ldg(&T72[__byte_perm(state[0].x,0,0x44443)])
			^ shared[5][__byte_perm(state[1].x,0,0x44443)];

	return_state[4] = __ldg(&T02[__byte_perm(state[7].y,0,0x44440)])
			^ shared[0][__byte_perm(state[6].y,0,0x44440)]
			^ __ldg(&T22[__byte_perm(state[5].y,0,0x44440)])
			^ shared[2][__byte_perm(state[4].y,0,0x44440)]
			^ shared[3][__byte_perm(state[3].y,0,0x44440)]
			^ __ldg(&T62[__byte_perm(state[1].y,0,0x44440)])
			^ shared[4][__byte_perm(state[2].y,0,0x44440)]
			^ __ldg(&T72[__byte_perm(state[0].y,0,0x44440)]);

	return_state[5] = __ldg(&T02[__byte_perm(state[7].y,0,0x44441)])
			^ shared[1][__byte_perm(state[5].y,0,0x44441)]
			^ __ldg(&T12[__byte_perm(state[6].y,0,0x44441)])
			^ shared[2][__byte_perm(state[4].y,0,0x44441)]
			^ shared[3][__byte_perm(state[3].y,0,0x44441)]
			^ shared[4][__byte_perm(state[2].y,0,0x44441)]
			^ __ldg(&T62[__byte_perm(state[1].y,0,0x44441)])
			^ __ldg(&T72[__byte_perm(state[0].y,0,0x44441)]);

	return_state[6] = __ldg(&T02[__byte_perm(state[7].y,0,0x44442)])
			^ shared[0][__byte_perm(state[6].y,0,0x44442)]
			^ shared[1][__byte_perm(state[5].y,0,0x44442)]
			^ shared[2][__byte_perm(state[4].y,0,0x44442)]
			^ shared[3][__byte_perm(state[3].y,0,0x44442)]
			^ shared[4][__byte_perm(state[2].y,0,0x44442)]
			^ __ldg(&T62[__byte_perm(state[1].y,0,0x44442)])
			^ __ldg(&T72[__byte_perm(state[0].y,0,0x44442)]);

	return_state[7] = __ldg(&T02[__byte_perm(state[7].y,0,0x44443)])
			^ __ldg(&T12[__byte_perm(state[6].y,0,0x44443)])
			^ shared[1][__byte_perm(state[5].y,0,0x44443)]
			^ shared[2][__byte_perm(state[4].y,0,0x44443)]
			^ shared[3][__byte_perm(state[3].y,0,0x44443)]
			^ shared[4][__byte_perm(state[2].y,0,0x44443)]
			^ __ldg(&T62[__byte_perm(state[1].y,0,0x44443)])
			^ __ldg(&T72[__byte_perm(state[0].y,0,0x44443)]);
}

__device__ __forceinline__
static void GOST_FS_LDG(const uint2 shared[6][256],const uint2 *const __restrict__ state, uint2* return_state) {

	return_state[0] = __ldg(&T02[__byte_perm(state[7].x,0,0x44440)])
			^ __ldg(&T12[__byte_perm(state[6].x,0,0x44440)])
			^ shared[1][__byte_perm(state[5].x,0,0x44440)]
			^ shared[2][__byte_perm(state[4].x,0,0x44440)]
			^ shared[3][__byte_perm(state[3].x,0,0x44440)]
			^ shared[4][__byte_perm(state[2].x,0,0x44440)]
			^ shared[5][__byte_perm(state[1].x,0,0x44440)]
			^ __ldg(&T72[__byte_perm(state[0].x,0,0x44440)]);

	return_state[1] = __ldg(&T02[__byte_perm(state[7].x,0,0x44441)])
			^ __ldg(&T12[__byte_perm(state[6].x,0,0x44441)])
			^ shared[1][__byte_perm(state[5].x,0,0x44441)]
			^ shared[2][__byte_perm(state[4].x,0,0x44441)]
			^ shared[3][__byte_perm(state[3].x,0,0x44441)]
			^ shared[4][__byte_perm(state[2].x,0,0x44441)]
			^ __ldg(&T72[__byte_perm(state[0].x,0,0x44441)])
			^ shared[5][__byte_perm(state[1].x,0,0x44441)];

	return_state[2] = __ldg(&T02[__byte_perm(state[7].x,0,0x44442)])
			^ __ldg(&T12[__byte_perm(state[6].x,0,0x44442)])
			^ shared[1][__byte_perm(state[5].x,0,0x44442)]
			^ shared[2][__byte_perm(state[4].x,0,0x44442)]
			^ shared[3][__byte_perm(state[3].x,0,0x44442)]
			^ shared[4][__byte_perm(state[2].x,0,0x44442)]
			^ shared[5][__byte_perm(state[1].x,0,0x44442)]
			^ __ldg(&T72[__byte_perm(state[0].x,0,0x44442)]);

	return_state[3] = __ldg(&T02[__byte_perm(state[7].x,0,0x44443)])
			^ __ldg(&T12[__byte_perm(state[6].x,0,0x44443)])
			^ shared[1][__byte_perm(state[5].x,0,0x44443)]
			^ shared[2][__byte_perm(state[4].x,0,0x44443)]
			^ shared[3][__byte_perm(state[3].x,0,0x44443)]
			^ shared[4][__byte_perm(state[2].x,0,0x44443)]
			^ shared[5][__byte_perm(state[1].x,0,0x44443)]
			^ __ldg(&T72[__byte_perm(state[0].x,0,0x44443)]);

	return_state[4] = __ldg(&T02[__byte_perm(state[7].y,0,0x44440)])
			^ shared[0][__byte_perm(state[6].y,0,0x44440)]
			^ __ldg(&T22[__byte_perm(state[5].y,0,0x44440)])
			^ shared[2][__byte_perm(state[4].y,0,0x44440)]
			^ shared[3][__byte_perm(state[3].y,0,0x44440)]
			^ shared[4][__byte_perm(state[2].y,0,0x44440)]
			^ __ldg(&T72[__byte_perm(state[0].y,0,0x44440)])
			^ __ldg(&T62[__byte_perm(state[1].y,0,0x44440)]);

	return_state[5] = __ldg(&T02[__byte_perm(state[7].y,0,0x44441)])
			^ __ldg(&T12[__byte_perm(state[6].y,0,0x44441)])
			^ shared[1][__byte_perm(state[5].y,0,0x44441)]
			^ shared[2][__byte_perm(state[4].y,0,0x44441)]
			^ shared[3][__byte_perm(state[3].y,0,0x44441)]
			^ shared[4][__byte_perm(state[2].y,0,0x44441)]
			^ __ldg(&T72[__byte_perm(state[0].y,0,0x44441)])
			^ __ldg(&T62[__byte_perm(state[1].y,0,0x44441)]);

	return_state[6] = __ldg(&T02[__byte_perm(state[7].y,0,0x44442)])
			^ __ldg(&T12[__byte_perm(state[6].y,0,0x44442)])
			^ __ldg(&T22[__byte_perm(state[5].y,0,0x44442)])
			^ shared[2][__byte_perm(state[4].y,0,0x44442)]
			^ shared[3][__byte_perm(state[3].y,0,0x44442)]
			^ shared[4][__byte_perm(state[2].y,0,0x44442)]
			^ __ldg(&T72[__byte_perm(state[0].y,0,0x44442)])
			^ __ldg(&T62[__byte_perm(state[1].y,0,0x44442)]);

	return_state[7] = __ldg(&T02[__byte_perm(state[7].y,0,0x44443)])
			^ shared[0][__byte_perm(state[6].y,0,0x44443)]
			^ __ldg(&T22[__byte_perm(state[5].y,0,0x44443)])
			^ shared[2][__byte_perm(state[4].y,0,0x44443)]
			^ shared[3][__byte_perm(state[3].y,0,0x44443)]
			^ shared[4][__byte_perm(state[2].y,0,0x44443)]
			^ __ldg(&T72[__byte_perm(state[0].y,0,0x44443)])
			^ __ldg(&T62[__byte_perm(state[1].y,0,0x44443)]);
}

__device__ __forceinline__
static void GOST_E12(const uint2 shared[6][256],uint2 *const __restrict__ K, uint2 *const __restrict__ state)
{
	uint2 t[8];
	for(int i=0; i<12; i++){
		GOST_FS(shared, state, t);

		#pragma unroll 8
		for(int j=0;j<8;j++)
			K[ j] ^= *(uint2*)&CC[i][j];

		#pragma unroll 8
		for(int j=0;j<8;j++)
			state[ j] = t[ j];

		GOST_FS_LDG(shared, K, t);

		#pragma unroll 8
		for(int j=0;j<8;j++)
			state[ j]^= t[ j];

		#pragma unroll 8
		for(int j=0;j<8;j++)
			K[ j] = t[ j];
	}
}

#pragma endregion


#pragma region Echo512_macros

#if PHI2BRANCH_TPB64 == 64
#define AES_GPU_INIT(sharedMemory) aes_gpu_init64(sharedMemory)
#elif PHI2BRANCH_TPB64 == 128
#define AES_GPU_INIT(sharedMemory) aes_gpu_init128(sharedMemory)
#elif PHI2BRANCH_TPB64 == 256
#define AES_GPU_INIT(sharedMemory) aes_gpu_init256(sharedMemory)
#elif PHI2BRANCH_TPB64 < 256
#define AES_GPU_INIT(sharedMemory) aes_gpu_init_lt_256(sharedMemory)
#else
#define AES_GPU_INIT(sharedMemory) aes_gpu_init_mt_256(sharedMemory)
#endif

#define ECHO_AES_SHIFT_IDX(i) ((((4 + (i >> 2) - (i % 4)) % 4) * 4 + (i % 4)) << 2)

#pragma endregion

#pragma region Echo512_inlines

__device__ __forceinline__
static void AES_2ROUND_MOVE(const uint32_t sharedMemory[4][256],
    uint32_t x0, uint32_t x1, uint32_t x2, uint32_t x3,
    uint32_t &k0,
    uint32_t &z0, uint32_t &z1, uint32_t &z2, uint32_t &z3) {
    uint32_t y0, y1, y2, y3;

    aes_round(sharedMemory,
        x0, x1, x2, x3,
        k0,
        y0, y1, y2, y3);

    aes_round(sharedMemory,
        y0, y1, y2, y3,
        z0, z1, z2, z3);

    k0++;
}

__device__ __forceinline__
static void AES_2ROUND_MOVE_CRIPPLE_23(const uint32_t sharedMemory[4][256],
    uint32_t x0, uint32_t x1, uint32_t x2, uint32_t x3,
    uint32_t &k0,
    uint32_t &z2, uint32_t &z3) {
    uint32_t y0, y1, y2, y3;

    aes_round(sharedMemory,
        x0, x1, x2, x3,
        k0,
        y0, y1, y2, y3);

    aes_round2_cripple_23(sharedMemory,
        y0, y1, y2, y3,
        z2, z3);

    k0++;
}

__device__ __forceinline__
void first_echo_round_64(const uint32_t sharedMemory[4][256], uint32_t* const paddedMsg, uint32_t* W, uint32_t &k0) {

    const uint32_t precalc_start_64_0[8] = { 0xe7e9f5f5, 0x14b8a457, 0xdbfde1dd, 0x9ac2dea3, 0x65978b09, 0xa4213d7e, 0x265f4382, 0x34514d9e };
    const uint32_t precalc_start_xx_1 = 0xf5e7e9f5;
    const uint32_t precalc_start_xx_2 = 0xb3b36b23;
    const uint32_t precalc_start_xx_3 = 0xb3dbe7af;
    const uint32_t precalc_end_64_12[16] = {
        0xb134347e, 0xea6f7e7e, 0xbd7731bd, 0x8a8a1968,
        0x579f9f33, 0xfbfbfbfb, 0xfbfbfbfb, 0xefefd3c7,
        0x2cb6b661, 0x6b23b3b3, 0xcf93a7cf, 0x9d9d3751,
        0x01425eb8, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
    };

    #pragma unroll 8
    for (int i = 0; i < 8; i++) {
        uint32_t idxWprime = ECHO_AES_SHIFT_IDX(i);
        W[idxWprime] = precalc_start_64_0[i];
        W[idxWprime + 1] = precalc_start_xx_1;
        W[idxWprime + 2] = precalc_start_xx_2;
        W[idxWprime + 3] = precalc_start_xx_3;
    }
    k0 += 8;

    #pragma unroll
    for (int i = 8; i < 12; i++) {
        uint32_t idxWprime = ECHO_AES_SHIFT_IDX(i);
        AES_2ROUND_MOVE(sharedMemory, paddedMsg[(i - 8 << 2)], paddedMsg[(i - 8 << 2) + 1], paddedMsg[(i - 8 << 2) + 2], paddedMsg[(i - 8 << 2) + 3],
            k0, W[idxWprime], W[idxWprime + 1], W[idxWprime + 2], W[idxWprime + 3]);
    }
    #pragma unroll
    for (int i = 12; i < 16; i++) {
        uint32_t idxWprime = ECHO_AES_SHIFT_IDX(i);
        W[idxWprime] = precalc_end_64_12[((i-12)<<2)];
        W[idxWprime + 1] = precalc_end_64_12[((i-12)<<2) + 1];
        W[idxWprime + 2] = precalc_end_64_12[((i-12)<<2) + 2];
        W[idxWprime + 3] = precalc_end_64_12[((i-12)<<2) + 3];
    }
    k0 += 4;

    // Mix Columns
    #pragma unroll 4
    for (int i = 0; i < 4; i++) {
        #pragma unroll 4
        for (int idx = 0; idx < 64; idx += 16) {
            uint32_t a = W[idx + i];
            uint32_t b = W[idx + i + 4];
            uint32_t c = W[idx + i + 8];
            uint32_t d = W[idx + i + 12];

            uint32_t ab = a ^ b;
            uint32_t bc = b ^ c;
            uint32_t cd = c ^ d;

            uint32_t t = (ab & 0x80808080);
            uint32_t t2 = (bc & 0x80808080);
            uint32_t t3 = (cd & 0x80808080);

            uint32_t abx = (t >> 7) * 27U ^ ((ab^t) << 1);
            uint32_t bcx = (t2 >> 7) * 27U ^ ((bc^t2) << 1);
            uint32_t cdx = (t3 >> 7) * 27U ^ ((cd^t3) << 1);

            W[idx + i] = bc ^ d ^ abx;
            W[idx + i + 4] = a ^ cd ^ bcx;
            W[idx + i + 8] = ab ^ d ^ cdx;
            W[idx + i + 12] = ab ^ c ^ (abx ^ bcx ^ cdx);
        }
    }
}

__device__ __forceinline__
void first_echo_round_80(const uint32_t sharedMemory[4][256], uint32_t* const paddedMsg, uint32_t* W, uint32_t &k0) {

    const uint32_t precalc_start_80_0[8] = { 0xc2031f3a, 0x428a9633, 0xe2eaf6f3, 0xc9f3efc1, 0x56869a2b, 0x789c801f, 0x81cbd7b1, 0x4a7b67ca };
    const uint32_t precalc_start_xx_1 = 0xf5e7e9f5;
    const uint32_t precalc_start_xx_2 = 0xb3b36b23;
    const uint32_t precalc_start_xx_3 = 0xb3dbe7af;
    const uint32_t precalc_end_80_13[12] = {
        0x83d3d3ab, 0xea6f7e7e, 0xbd7731bd, 0x8a8a1968,
        0x5d99993f, 0x6b23b3b3, 0xcf93a7cf, 0x9d9d3751,
        0x57706cdc, 0xe4736c70, 0xf53fa165, 0xd6be2d00
    };

    #pragma unroll
    for (int i = 0; i < 8; i++) {
        uint32_t idxWprime = ECHO_AES_SHIFT_IDX(i);
        W[idxWprime] = precalc_start_80_0[i];
        W[idxWprime + 1] = precalc_start_xx_1;
        W[idxWprime + 2] = precalc_start_xx_2;
        W[idxWprime + 3] = precalc_start_xx_3;
    }
    k0 += 8;

    #pragma unroll
    for (int i = 8; i < 13; i++) {
        uint32_t idxWprime = ECHO_AES_SHIFT_IDX(i);
        AES_2ROUND_MOVE(sharedMemory, paddedMsg[(i - 8 << 2)], paddedMsg[(i - 8 << 2) + 1], paddedMsg[(i - 8 << 2) + 2], paddedMsg[(i - 8 << 2) + 3],
            k0, W[idxWprime], W[idxWprime + 1], W[idxWprime + 2], W[idxWprime + 3]);
    }

    #pragma unroll
    for (int i = 13; i < 16; i++) {
        uint32_t idxWprime = ECHO_AES_SHIFT_IDX(i);
        W[idxWprime] = precalc_end_80_13[((i-13)<<2)];
        W[idxWprime + 1] = precalc_end_80_13[((i-13)<<2) + 1];
        W[idxWprime + 2] = precalc_end_80_13[((i-13)<<2) + 2];
        W[idxWprime + 3] = precalc_end_80_13[((i-13)<<2) + 3];
    }
    k0 += 3;


    // Mix Columns
    #pragma unroll 4
    for (int i = 0; i < 4; i++) {
        #pragma unroll 4
        for (int idx = 0; idx < 64; idx += 16) {
            uint32_t a = W[idx + i];
            uint32_t b = W[idx + i + 4];
            uint32_t c = W[idx + i + 8];
            uint32_t d = W[idx + i + 12];

            uint32_t ab = a ^ b;
            uint32_t bc = b ^ c;
            uint32_t cd = c ^ d;

            uint32_t t = (ab & 0x80808080);
            uint32_t t2 = (bc & 0x80808080);
            uint32_t t3 = (cd & 0x80808080);

            uint32_t abx = (t >> 7) * 27U ^ ((ab^t) << 1);
            uint32_t bcx = (t2 >> 7) * 27U ^ ((bc^t2) << 1);
            uint32_t cdx = (t3 >> 7) * 27U ^ ((cd^t3) << 1);

            W[idx + i] = bc ^ d ^ abx;
            W[idx + i + 4] = a ^ cd ^ bcx;
            W[idx + i + 8] = ab ^ d ^ cdx;
            W[idx + i + 12] = ab ^ c ^ (abx ^ bcx ^ cdx);
        }
    }
}

__device__ __forceinline__
void last_echo_round(const uint32_t sharedMemory[4][256], uint32_t *W, uint32_t &k0) {
    // BIG.SubWords
    // Are needed for next ShiftRows : W0, W2, W5, W7, W8, W10, W13, W15
    // All others will be short-circuit but k still need to be incremented
    // BIG.ShiftRows
    // W0 to W3 and W8 to W11 only are needed for next Mix columns
    // W0 and W8 don't shift
    // W1 and W9 comes from right neighbour (ie W5 and W13)
    // W'[4..7] = W[20..23]; W'[36..39] = W[52..55];
    // W2 and W10 come from 2 columns away (ie swapped)
    // W'[8..11] = W[40..43]; W'[40..43] = W[8..11];
    // W3 and W11 comes from left neighbour (ie W15 and W7)
    // W'[12..15] = W[60..63]; W'[44..47] = W[28..31];

    // W0 not moving
    AES_2ROUND(sharedMemory, W[0], W[1], W[2], W[3], k0);
    k0++;
    // W2 go to W10 (swapped)
    uint32_t t0, t1, t2, t3;
    t0 = W[40]; t1 = W[41]; t2 = W[42]; t3 = W[43];
    AES_2ROUND_MOVE(sharedMemory, W[8], W[9], W[10], W[11], k0, W[40], W[41], W[42], W[43]);
    k0 += 2;
    // W5 go to W1
    AES_2ROUND_MOVE(sharedMemory, W[20], W[21], W[22], W[23], k0, W[4], W[5], W[6], W[7]);
    k0++;
    // W7 go to W11
    AES_2ROUND_MOVE(sharedMemory, W[28], W[29], W[30], W[31], k0, W[44], W[45], W[46], W[47]);
    // W8 not moving
    AES_2ROUND(sharedMemory, W[32], W[33], W[34], W[35], k0);
    k0++;
    // W10 go to W2 (swapped)
    AES_2ROUND_MOVE(sharedMemory, t0, t1, t2, t3, k0, W[8], W[9], W[10], W[11]);
    k0 += 2;
    // W13 go to W9
    AES_2ROUND_MOVE(sharedMemory, W[52], W[53], W[54], W[55], k0, W[36], W[37], W[38], W[39]);
    k0++;
    // W15 go to W3
    AES_2ROUND_MOVE(sharedMemory, W[60], W[61], W[62], W[63], k0, W[12], W[13], W[14], W[15]);

    // BIG.MixColumns
    // For Echo-512 Hsize, we only need to calculate W[0..15] (W0 to W3) and W[32..47] (W8 to W11)
    #pragma unroll 4
    for (int i = 0; i < 4; i++) {
        #pragma unroll 2
        for (int idx = 0; idx < 64; idx += 32) {
            uint32_t a = W[idx + i];
            uint32_t b = W[idx + i + 4];
            uint32_t c = W[idx + i + 8];
            uint32_t d = W[idx + i + 12];

            uint32_t ab = a ^ b;
            uint32_t bc = b ^ c;
            uint32_t cd = c ^ d;

            uint32_t t = (ab & 0x80808080);
            uint32_t t2 = (bc & 0x80808080);
            uint32_t t3 = (cd & 0x80808080);

            uint32_t abx = (t >> 7) * 27U ^ ((ab^t) << 1);
            uint32_t bcx = (t2 >> 7) * 27U ^ ((bc^t2) << 1);
            uint32_t cdx = (t3 >> 7) * 27U ^ ((cd^t3) << 1);

            W[idx + i] = bc ^ d ^ abx;
            W[idx + i + 4] = a ^ cd ^ bcx;
            W[idx + i + 8] = ab ^ d ^ cdx;
            W[idx + i + 12] = ab ^ c ^ (abx ^ bcx ^ cdx);
        }
    }
}

__device__ __forceinline__
void last_echo_round_final(const uint32_t sharedMemory[4][256], uint32_t *W, uint32_t &k0) {
    // BIG.SubWords
    // Are needed for next ShiftRows : W0, W2, W5, W7, W8, W10, W13, W15
    // All others will be short-circuit but k still need to be incremented
    // BIG.ShiftRows
    // W0 to W3 and W8 to W11 only are needed for next Mix columns
    // W0 and W8 don't shift
    // W1 and W9 comes from right neighbour (ie W5 and W13)
    // W'[4..7] = W[20..23]; W'[36..39] = W[52..55];
    // W2 and W10 come from 2 columns away (ie swapped)
    // W'[8..11] = W[40..43]; W'[40..43] = W[8..11];
    // W3 and W11 comes from left neighbour (ie W15 and W7)
    // W'[12..15] = W[60..63]; W'[44..47] = W[28..31];

    // W0 not moving
    //AES_2ROUND(sharedMemory, W[0], W[1], W[2], W[3], k0);
    AES_2ROUND_MOVE_CRIPPLE_23(sharedMemory, W[0], W[1], W[2], W[3], k0, W[2], W[3]);
    k0++;
    // W2 go to W10 (swapped)
    uint32_t t0, t1, t2, t3;
    t0 = W[40]; t1 = W[41]; t2 = W[42]; t3 = W[43];
    //AES_2ROUND_MOVE(sharedMemory, W[8], W[9], W[10], W[11], k0, W[40], W[41], W[42], W[43]);
    AES_2ROUND_MOVE_CRIPPLE_23(sharedMemory, W[8], W[9], W[10], W[11], k0, W[42], W[43]);
    k0 += 2;
    // W5 go to W1
    //AES_2ROUND_MOVE(sharedMemory, W[20], W[21], W[22], W[23], k0, W[4], W[5], W[6], W[7]);
    AES_2ROUND_MOVE_CRIPPLE_23(sharedMemory, W[20], W[21], W[22], W[23], k0, W[6], W[7]);
    k0++;
    // W7 go to W11
    //AES_2ROUND_MOVE(sharedMemory, W[28], W[29], W[30], W[31], k0, W[44], W[45], W[46], W[47]);
    AES_2ROUND_MOVE_CRIPPLE_23(sharedMemory, W[28], W[29], W[30], W[31], k0, W[46], W[47]);
    // W8 not moving
    //AES_2ROUND(sharedMemory, W[32], W[33], W[34], W[35], k0);
    AES_2ROUND_MOVE_CRIPPLE_23(sharedMemory, W[32], W[33], W[34], W[35], k0, W[34], W[35]);
    k0++;
    // W10 go to W2 (swapped)
    //AES_2ROUND_MOVE(sharedMemory, t0, t1, t2, t3, k0, W[8], W[9], W[10], W[11]);
    AES_2ROUND_MOVE_CRIPPLE_23(sharedMemory, t0, t1, t2, t3, k0, W[10], W[11]);
    k0 += 2;
    // W13 go to W9
    //AES_2ROUND_MOVE(sharedMemory, W[52], W[53], W[54], W[55], k0, W[36], W[37], W[38], W[39]);
    AES_2ROUND_MOVE_CRIPPLE_23(sharedMemory, W[52], W[53], W[54], W[55], k0, W[38], W[39]);
    k0++;
    // W15 go to W3
    //AES_2ROUND_MOVE(sharedMemory, W[60], W[61], W[62], W[63], k0, W[12], W[13], W[14], W[15]);
    aes_round(sharedMemory, W[60], W[61], W[62], W[63], k0, t0, t1, t2, t3);
    aes_round2_cripple_23(sharedMemory, t0, t1, t2, t3, W[14], W[15]);

    // BIG.MixColumns
    // For Echo-512 final, we only need to calculate W[6], W[7], W[38], W[39]
    uint32_t bc = W[6] ^ W[10];
    uint32_t tmp = (bc & 0x80808080);
    uint32_t bcx = (tmp >> 7) * 27U ^ ((bc^tmp) << 1);
    W[6] = W[2] ^ W[10] ^ W[14] ^ bcx;

    bc = W[7] ^ W[11];
    tmp = (bc & 0x80808080);
    bcx = (tmp >> 7) * 27U ^ ((bc^tmp) << 1);
    W[7] = W[3] ^ W[11] ^ W[15] ^ bcx;

    bc = W[38] ^ W[42];
    tmp = (bc & 0x80808080);
    bcx = (tmp >> 7) * 27U ^ ((bc^tmp) << 1);
    W[38] = W[34] ^ W[42] ^ W[46] ^ bcx;

    bc = W[39] ^ W[43];
    tmp = (bc & 0x80808080);
    bcx = (tmp >> 7) * 27U ^ ((bc^tmp) << 1);
    W[39] = W[35] ^ W[43] ^ W[47] ^ bcx;

}

__device__
static void echo_round(const uint32_t sharedMemory[4][256], uint32_t *W, uint32_t &k0) {

    // Big Sub Words
    #pragma unroll 16
    for (int idx = 0; idx < 16; idx++) {
        AES_2ROUND(sharedMemory, W[(idx << 2) + 0], W[(idx << 2) + 1], W[(idx << 2) + 2], W[(idx << 2) + 3], k0);
    }

    // Shift Rows
    #pragma unroll 4
    for (int i = 0; i < 4; i++) {
        uint32_t t[4];
        t[0] = W[i + 4];
        t[1] = W[i + 8];
        t[2] = W[i + 24];
        t[3] = W[i + 60];

        W[i + 4] = W[i + 20];
        W[i + 8] = W[i + 40];
        W[i + 24] = W[i + 56];
        W[i + 60] = W[i + 44];

        W[i + 20] = W[i + 36];
        W[i + 40] = t[1];
        W[i + 56] = t[2];
        W[i + 44] = W[i + 28];

        W[i + 28] = W[i + 12];
        W[i + 12] = t[3];
        W[i + 36] = W[i + 52];
        W[i + 52] = t[0];
    }

    // Mix Columns
    #pragma unroll 4
    for (int i = 0; i < 4; i++) {
        #pragma unroll 4
        for (int idx = 0; idx < 64; idx += 16) {
            uint32_t a[4];
            a[0] = W[idx + i];
            a[1] = W[idx + i + 4];
            a[2] = W[idx + i + 8];
            a[3] = W[idx + i + 12];

            uint32_t ab = a[0] ^ a[1];
            uint32_t bc = a[1] ^ a[2];
            uint32_t cd = a[2] ^ a[3];

            uint32_t t, t2, t3;
            t = (ab & 0x80808080);
            t2 = (bc & 0x80808080);
            t3 = (cd & 0x80808080);

            uint32_t abx = (t >> 7) * 27U ^ ((ab^t) << 1);
            uint32_t bcx = (t2 >> 7) * 27U ^ ((bc^t2) << 1);
            uint32_t cdx = (t3 >> 7) * 27U ^ ((cd^t3) << 1);

            W[idx + i] = bc ^ a[3] ^ abx;
            W[idx + i + 4] = a[0] ^ cd ^ bcx;
            W[idx + i + 8] = ab ^ a[3] ^ cdx;
            W[idx + i + 12] = ab ^ a[2] ^ (abx ^ bcx ^ cdx);
        }
    }

}

#pragma endregion



#pragma region Echo512_64



__global__
__launch_bounds__(PHI2BRANCH_TPB64, 3)
void cuda_phi2_streeb_echo512_gpu_hash_64(const uint32_t threads, uint32_t* g_hash) {

   	__shared__ uint2 streebogShared[6][256]; //streebogShared[8][256]; crop unused to fit 12 warps
    __shared__ uint32_t echoShared[4][256];

    LUT_GPU_INIT(streebogShared, threadIdx.x)
    AES_GPU_INIT(echoShared);

    __threadfence_block();

    uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads) {

        uint32_t *pHash = &g_hash[thread<<4];

        if (pHash[0] & 1) { // Single streebog round

            uint2 buf[8], t[8], temp[8], K0[8], hash[8];

	        *(uint2x4*)&hash[0] = __ldg4((uint2x4*)&pHash[0]);
	        *(uint2x4*)&hash[4] = __ldg4((uint2x4*)&pHash[8]);

	        K0[0] = vectorize(0x74a5d4ce2efc83b3);

	        #pragma unroll 8
	        for(int i=0;i<8;i++) {
		        buf[ i] = K0[ 0] ^ hash[ i];
	        }

	        for(int i=0; i<12; i++) {
		        GOST_FS(streebogShared, buf, temp);
		        #pragma unroll 8
		        for(uint32_t j=0;j<8;j++){
			        buf[ j] = temp[ j] ^ *(uint2*)&precomputed_values[i][j];
		        }
	        }
	        #pragma unroll 8
	        for(int j=0;j<8;j++) {
		        buf[ j] ^= hash[ j];
	        }
	        #pragma unroll 8
	        for(int j=0;j<8;j++) {
		        K0[ j] = buf[ j];
	        }

	        K0[7].y ^= 0x00020000;

	        GOST_FS(streebogShared, K0, t);

	        #pragma unroll 8
	        for(int i=0;i<8;i++)
		        K0[ i] = t[ i];

	        t[7].y ^= 0x01000000;

	        GOST_E12(streebogShared, K0, t);

	        #pragma unroll 8
	        for(int j=0;j<8;j++)
		        buf[ j] ^= t[ j];

	        buf[7].y ^= 0x01000000;

	        GOST_FS(streebogShared, buf,K0);

	        buf[7].y ^= 0x00020000;

	        #pragma unroll 8
	        for(int j=0;j<8;j++)
		        t[ j] = K0[ j];

	        t[7].y ^= 0x00020000;

	        GOST_E12(streebogShared, K0, t);

	        #pragma unroll 8
	        for(int j=0;j<8;j++)
		        buf[ j] ^= t[ j];

	        GOST_FS(streebogShared, buf,K0); // K = F(h)

	        hash[7]+= vectorize(0x0100000000000000);

	        #pragma unroll 8
	        for(int j=0;j<8;j++)
		        t[ j] = K0[ j] ^ hash[ j];

	        GOST_E12(streebogShared, K0, t);

	        *(uint2x4*)&pHash[0] = *(uint2x4*)&t[0] ^ *(uint2x4*)&hash[0] ^ *(uint2x4*)&buf[0];
	        *(uint2x4*)&pHash[8] = *(uint2x4*)&t[4] ^ *(uint2x4*)&hash[4] ^ *(uint2x4*)&buf[4];


        } else { // Dual echo-512 hash

            uint32_t inMsg[16];
            uint32_t k0 = 512;
            uint32_t W[64]; // 128bits 4x4 State matrix

            *(uint2x4*)&inMsg[0] = __ldg4((uint2x4*)&pHash[0]);
            *(uint2x4*)&inMsg[8] = __ldg4((uint2x4*)&pHash[8]);

            first_echo_round_64(echoShared, inMsg, W, k0);
            for (int i = 0; i < 8; i++) 
                echo_round(echoShared, W, k0);
            last_echo_round(echoShared, W, k0);

            // BIG.final
            #pragma unroll 4
            for (int i = 0; i < 16; i += 4) {
                W[i] ^= 512U ^ inMsg[i] ^ W[i + 32];
                W[i + 1] ^= inMsg[i + 1] ^ W[i + 33];
                W[i + 2] ^= inMsg[i + 2] ^ W[i + 34];
                W[i + 3] ^= inMsg[i + 3] ^ W[i + 35];
            }

            // Second Echo-512 hash
            k0 = 512;
            //uint32_t W[64]; // will be reinitialized at the beginning of first echo round

            #pragma unroll
            for (int padIdx = 0; padIdx < 16; padIdx ++)
                inMsg[padIdx] = W[padIdx];

            first_echo_round_64(echoShared, inMsg, W, k0);
            for (int i = 0; i < 8; i++) 
                echo_round(echoShared, W, k0);
            last_echo_round(echoShared, W, k0);

            // BIG.final
            #pragma unroll 4
            for (int i = 0; i < 16; i += 4) {
                W[i] ^= 512U ^ inMsg[i] ^ W[i + 32];
                W[i + 1] ^= inMsg[i + 1] ^ W[i + 33];
                W[i + 2] ^= inMsg[i + 2] ^ W[i + 34];
                W[i + 3] ^= inMsg[i + 3] ^ W[i + 35];
            }

            #pragma unroll
            for (int i = 0; i < 16; i += 2)
                AS_UINT2(&pHash[i]) = AS_UINT2(&W[i]);
        }

    }
}

__host__
void cuda_phi2_branhc_streeb_echo512_cpu_hash_64(const uint32_t threads, uint32_t *d_hash) {

    dim3 grid((threads + PHI2BRANCH_TPB64 - 1) / PHI2BRANCH_TPB64);
    dim3 block(PHI2BRANCH_TPB64);

    cuda_phi2_streeb_echo512_gpu_hash_64<<<grid, block>>>(threads, d_hash);
}

#pragma endregion

#undef AES_GPU_INIT
#undef LUT_GPU_INIT
